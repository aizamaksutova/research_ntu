#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void stress_pcie(int num_iterations, int transfer_size_bytes)
{
    char* src = new char[transfer_size_bytes];
    char* dst = new char[transfer_size_bytes];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < num_iterations; i++)
    {
        hipEventRecord(start);
        hipMemcpy(dst, src, transfer_size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(src, dst, transfer_size_bytes, hipMemcpyDeviceToHost);
        hipEventRecord(stop);
    }

    hipEventSynchronize(stop);
    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    printf("Time taken for %d iterations of %d-byte transfers: %f ms\n", num_iterations, transfer_size_bytes, elapsed_time_ms);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete[] src;
    delete[] dst;
}

int main()
{
    int num_iterations = 1000;
    int transfer_size_bytes = 1024 * 1024; // 1 MB

    stress_pcie<<<1, 1>>>(num_iterations, transfer_size_bytes);
    hipDeviceSynchronize();
}
